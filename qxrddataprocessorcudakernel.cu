
#include <hip/hip_runtime.h>
/******************************************************************
*
*  $Id: qxrddataprocessorcudakernel.cu,v 1.2 2010/09/13 19:59:35 jennings Exp $
*
*******************************************************************/

extern "C" void qxrdCuda();

void qxrdCuda()
{
}

#define BLOCK_SIZE  16
#define INPUT_OVERSAMPLE 1
#define OUTPUT_OVERSAMPLE 1
#define OUTPUT_BLOCK_SIZE (BLOCK_SIZE*2*OUTPUT_OVERSAMPLE)

__global__ void
    circularIntegrate(unsigned short *g_image, bool *g_mask, int *g_output, float xc, float yc)
{
  __shared__ int sums[OUTPUT_BLOCK_SIZE], npix[OUTPUT_BLOCK_SIZE];
  __shared__ int minrad;

  const int index = BLOCK_SIZE*threadIdx.y + threadIdx.x;

  if (index < OUTPUT_BLOCK_SIZE) {
    sums[index] = 0;
    npix[index] = 0;
  }

  const unsigned int blkminx = blockIdx.x * blockDim.x;
  const unsigned int blkminy = blockIdx.y * blockDim.y;
  const unsigned int blkmaxx = blkminx + BLOCK_SIZE;
  const unsigned int blkmaxy = blkminy + BLOCK_SIZE;

  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    if (xc < blkminx) {
      if (yc < blkminy) {
        minrad = (sqrt((blkminx - xc)*(blkminx - xc) + (blkminy - yc)*(blkminy - yc))*OUTPUT_OVERSAMPLE);
      } else if (yc <= blkmaxy) {
        minrad = (blkminx-xc)*OUTPUT_OVERSAMPLE;
      } else {
        minrad = (sqrt((blkminx - xc)*(blkminx - xc) + (blkmaxy - yc)*(blkmaxy - yc))*OUTPUT_OVERSAMPLE);
      }
    } else if (xc <= blkmaxx) {
      if (yc < blkminy) {
        minrad = (blkminy-yc)*OUTPUT_OVERSAMPLE;
      } else if (yc <= blkmaxy) {
        minrad = 0;
      } else {
        minrad = (yc - blkmaxy)*OUTPUT_OVERSAMPLE;
      }
    } else {
      if (yc < blkminy) {
        minrad = (sqrt((blkmaxx - xc)*(blkmaxx - xc) + (blkminy - yc)*(blkminy - yc))*OUTPUT_OVERSAMPLE);
      } else if (yc <= blkmaxy) {
        minrad = (xc - blkmaxx)*OUTPUT_OVERSAMPLE;
      } else {
        minrad = (sqrt((blkmaxx - xc)*(blkmaxx - xc) + (blkmaxy - yc)*(blkmaxy - yc))*OUTPUT_OVERSAMPLE);
      }
    }
  }

  __syncthreads();

  const unsigned int ix = blockIdx.x * blockDim.x + threadIdx.x;
  const unsigned int iy = blockIdx.y * blockDim.y + threadIdx.y;
  const unsigned int xdim = 2048;
  const unsigned int ydim = 2048;
  float ios = 0.5/((float) INPUT_OVERSAMPLE);
  if ((ix < xdim) && (iy < ydim)) {
    if (g_mask[iy*xdim + ix]) {
      for (int iysub = 0; iysub < INPUT_OVERSAMPLE; iysub++) {
        for (int ixsub = 0; ixsub < INPUT_OVERSAMPLE; ixsub++) {
          float dx = ios + (ix - xc) + ((float)ixsub)/((float)INPUT_OVERSAMPLE);
          float dy = ios + (iy - yc) + ((float)iysub)/((float)INPUT_OVERSAMPLE);
          float radius = sqrt(dx*dx + dy*dy);

          unsigned short value = g_image[iy*xdim + ix];

          int bin = radius*OUTPUT_OVERSAMPLE - minrad;

          if ((bin >= 0) && (bin < OUTPUT_BLOCK_SIZE)) {
            atomicAdd(&sums[bin], value);
            atomicAdd(&npix[bin], 1);
          }
        }
      }
    }
  }

  __syncthreads();

  const int outbin = threadIdx.y * BLOCK_SIZE + threadIdx.x;

  if (outbin < OUTPUT_BLOCK_SIZE) {
    if (npix[outbin] > 0) {
      atomicAdd(&g_output[2*(minrad+outbin)+0], sums[outbin]);
      atomicAdd(&g_output[2*(minrad+outbin)+1], npix[outbin]);
    }
  }
}

/******************************************************************
*
*  $Log: qxrddataprocessorcudakernel.cu,v $
*  Revision 1.2  2010/09/13 19:59:35  jennings
*  Merged
*
*  Revision 1.1.2.2  2010/06/16 19:22:39  jennings
*  First steps to adding CUDA support
*
*
*******************************************************************/

